#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#define size 4096
#define row 16
#define loop 256
//row is size/(32*sqrt(brock_num)) and loop % 2^2n == 0 loop = row^2
#define N 100
#define r 0.1
#define blocksize 1024
//size is a grid size and N is how many times to calculate
#define block_num 64

__global__
void calc(float *u1, float *u2, int n){
	int i2 = threadIdx.x;
	int i3 = blockIdx.x;
	int i4 = blockDim.x;


	for(int j = 0; j < loop; j++){
   	//int i = i2 + blockrow*(j%row) + (j/row) * blocksize * row;
   	int i = i2 + i4 * j + i4*loop*i3;
		i = i + ((i/size)*2) + 1 + (size + 2);
		if(n%2==0){
			float x1 = u1[i]; float x2 = u1[i+1]; float x3 = u1[i-1]; float x4 = u1[i+(size+2)]; float x5 = u1[i-(size+2)];
    	//__syncthreads();
    	u2[i] = (1 - 4*r) * x1 + r * (x2 + x3 + x4 + x5);
    	__syncthreads();	
		}
		else {
			float x1 = u2[i]; float x2 = u2[i+1]; float x3 = u2[i-1]; float x4 = u2[i+(size+2)]; float x5 = u2[i-(size+2)];
    	//__syncthreads();
    	u1[i] = (1 - 4*r) * x1 + r * (x2 + x3 + x4 + x5);
    	__syncthreads();
    }
	}
	return ;
}



int main(){

	float u_odd [(size+2)*(size+2)];
	float u_even [(size+2)*(size+2)];
	float u_result [(size+2)*(size+2)];

	float* u1;
	float* u2;

	for(int i = 0; i < (size+2)*(size+2); i++){
    u_even[i] = 0;
		u_odd[i] = 0;
		u_result[i] = 0;
  }

	//initialize u_even
	for(int i = 0; i < (size+2)*(size+2); i++){
		if(i % (size+2) != 0 && i % (size+2) != (size+1) && i/(size+2) != 0 && i/(size+2) != (size+1)){
			u_even[i] = 1;
		}
	}


	struct timeval t0,t1;

	const int csize = (size+2)*(size+2)*sizeof(float);
	hipMalloc((void**)&u1,csize);
	hipMalloc((void**)&u2,csize);
	if (N%2 == 0){
		hipMemcpy(u1,u_even,csize,hipMemcpyHostToDevice);
		hipMemcpy(u2,u_odd,csize,hipMemcpyHostToDevice);
	}else{
		hipMemcpy(u2,u_even,csize,hipMemcpyHostToDevice);
    hipMemcpy(u1,u_odd,csize,hipMemcpyHostToDevice);
	}
	dim3 dimBlock(blocksize,1);
	dim3 dimGrid(block_num,1);
	//時間ぶんloopする
  gettimeofday(&t0, NULL);
	for (int i = N; i > 0; i--){
		calc<<<dimGrid,dimBlock>>>(u1,u2,i);
		hipDeviceSynchronize();
	}
	gettimeofday(&t1, NULL);
	hipMemcpy(u_result,u1,csize,hipMemcpyDeviceToHost);
	hipFree(u1);
	hipFree(u2);
	
	printf("Elapsed time = %lf\n",((double)(t1.tv_sec - t0.tv_sec) + (double)(t1.tv_usec - t0.tv_usec)*1.0e-6));

/*
	for(int i = 0; i < (size+2)*(size+2); i++){
    if (i % (size+2) != 0 && i % (size+2) != (size+1) && i/(size+2) != 0 && i/(size+2) != (size+1)){
		printf("%.3f ",u_result[i]);
		}
		if(i % (size+2) == (size+1)){
			printf("\n");
		}
  }
*/
	
  return EXIT_SUCCESS;
}
